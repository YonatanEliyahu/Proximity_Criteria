#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "myProto.h"
#include "myStructs.h"
#define THREADS 384

__global__ void calculatePoint(Axis *axisArr, Point *pointArr, int numElements, double t)
{ // the following function will calcualte the point location in the specified t
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        pointArr[i].x = ((axisArr[i].x2 - axisArr[i].x1) / 2) * sin(t * M_PI / 2) + (axisArr[i].x2 + axisArr[i].x1) / 2;
        pointArr[i].y = axisArr[i].a * pointArr[i].x + axisArr[i].b;
    }
}
__global__ void SingularProximityCriteria(int index, int *d_temp, Point *pointArr, int N, float D, int K)
{
    // the following function will check if there are K points in the radious of the checked point and set the flag pointer to 1/0 acording to the results
    *d_temp = 0;
    int counter = 0;
    Point p1 = pointArr[index];                // the point we are checking // used for readability
    for (int i = 0; i < N && counter < K; i++) // as long as we didnt go through all the point or we havn't found K points in the radious
    {
        if (index == i)
            continue;
        Point p2 = pointArr[i]; // other point // used for readability
        if (sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2)) <= D)
            counter++;
    }
    if (counter == K)
        *d_temp = 1;
}

__global__ void ProximityCriteria(int rank, int chunkSize, int *flags, Point *pointArr, int numElements, float D, int K)
{
    // the following function will check if there are K points in the radious of the checked point and will set a flag in the flagArr
    int tid = blockDim.x * blockIdx.x + threadIdx.x; // relative index
    int index = tid + rank * chunkSize;              // acluat index
    int counter = 0;
    if (index < numElements)
    {
        Point p1 = pointArr[index];                          // the point we are checking // used for readability
        for (int i = 0; i < numElements && counter < K; i++) // as long as we didnt go through all the point or we havn't found K points in the radious
        {
            if (index == i)
                continue;
            Point p2 = pointArr[i]; // other point // used for readability
            if (sqrt(pow(p1.x - p2.x, 2) + pow(p1.y - p2.y, 2)) <= D)
                counter++;
        }
        if (counter == K)
            flags[tid] = 1;
    }
}

int computePointsOnGPU(Axis *axisArr, Point *pointArr, int numElements, double t)
{
    /* the following function will get sub arr of axises
        and compute the location of the points in the current t
    */
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // allocate arr for the axises
    size_t size = numElements * sizeof(Axis);
    Axis *d_Axis;
    err = hipMalloc((void **)&d_Axis, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Axis, axisArr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // allocate arr for the points
    size = numElements * sizeof(Point);
    Point *d_Points;
    err = hipMalloc((void **)&d_Points, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Kernel
    int threadsPerBlock = THREADS;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;            // create enough blocks for all the points
    calculatePoint<<<blocksPerGrid, threadsPerBlock>>>(d_Axis, d_Points, numElements, t); // compute points
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from GPU to the host memory
    err = hipMemcpy(pointArr, d_Points, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy data from device to host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_Axis);
    hipFree(d_Points);
    return 1;
}

int checkLastHits(Point *allPoints, int N, int *globalFlags, float D, int K)
{
    /*the following function will calculate the distance between the last hits and the rest of the points,
    if there are K points in a radious of a spesific point (in distancec of D),
    the point will satesfy the Proximity Criteria and will be save in flags arr
    the function will return indicator if those point are enough and the Proximity Criteria is satesfied
    */
    hipError_t err = hipSuccess;
    // allocate arr for the points
    size_t size = N * sizeof(Point);
    Point *d_Points;
    err = hipMalloc((void **)&d_Points, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Points, allPoints, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        hipFree(d_Points);
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int counter = 0;
    int *d_temp;
    err = hipMalloc((void **)&d_temp, sizeof(int));
    for (int i = 0; i < N && counter < 3; i++) // stops when hits three points that satesfy the Proximity Criteria
    {
        if (globalFlags[i])
        {
            SingularProximityCriteria<<<1, 1>>>(i, d_temp, d_Points, N, D, K); // set singular flag
            hipMemcpy(&globalFlags[i], d_temp, sizeof(int), hipMemcpyDeviceToHost);
            if (globalFlags[i])
                counter++;
        }
    }
    hipFree(d_Points);
    hipFree(d_temp);
    return counter;
}

void checkProximityCriteriaOnGPU(int rank, Point *allPoints, int N, int *flags, int chunkSize, float D, int K)
{
    /*the following function will calculate the distance between all the points,
    if there are K points in a radious of a spesific point (in distancec of D),
    the point will satesfy the Proximity Criteria and will be save in flags arr (the result of the function)
    */

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // allocate arr for the points
    size_t size = N * sizeof(Point);
    Point *d_Points;
    err = hipMalloc((void **)&d_Points, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copy data from host to the GPU memory
    err = hipMemcpy(d_Points, allPoints, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        hipFree(d_Points);
        fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Defining flag arr that will hold the points that answer the Proximity Criteria
    int *d_Flags;
    size = chunkSize * sizeof(int);
    err = hipMalloc((void **)&d_Flags, size);
    if (err != hipSuccess)
    {
        hipFree(d_Flags);
        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemset(d_Flags, 0, size);
    if (err != hipSuccess)
    {
        hipFree(d_Flags);
        hipFree(d_Points);
        fprintf(stderr, "Failed to set device memory to zero- %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Launch the Kernel
    int threadsPerBlock = THREADS;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;                                    // create enough blocks for all the points
    ProximityCriteria<<<blocksPerGrid, threadsPerBlock>>>(rank, chunkSize, d_Flags, d_Points, N, D, K); // set flagArr

    err = hipMemcpy(flags, d_Flags, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        hipFree(d_Flags);
        hipFree(d_Points);
        free(flags);
        fprintf(stderr, "Failed to copy data from device to host - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    hipFree(d_Flags);
    hipFree(d_Points);
}
